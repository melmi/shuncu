#include "hip/hip_runtime.h"
#include "kernels.h"

void HandleError(hipError_t err, const char *file, int line) 
{
	if (err != hipSuccess) 
	{
		printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
		exit(EXIT_FAILURE);
	}
}

void HandleError( hipError_t err ) 
{
	HandleError( err, __FILE__, __LINE__ );
}

void init_device_mesh(device_mesh& dm, mesh m)
{
	dm.ne  = m.ne;
	dm.nv  = m.nv;
	dm.ns  = m.ns;
	dm.nbv = m.nbv;
	dm.nbs = m.nbs;

	HandleError(hipMalloc((void**)&dm.a , m.nv * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.dl, m.ns * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.dn, m.ns * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.nx, m.ns * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.ny, m.ns * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.v1, m.ns * sizeof(int  )));
	HandleError(hipMalloc((void**)&dm.v2, m.ns * sizeof(int  )));
	HandleError(hipMalloc((void**)&dm.ibc, m.ns * sizeof(int  )));
	
	HandleError(hipMalloc((void**)&dm.nbx, m.nbs * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.nby, m.nbs * sizeof(float)));

	HandleError(hipMalloc((void**)&dm.bsides  , m.nbs * sizeof(int)));
	HandleError(hipMalloc((void**)&dm.bvertices, m.nbv * sizeof(int)));

	HandleError(hipMalloc((void**)&dm.px, m.nv * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.py, m.nv * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.ux, m.nv * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.uy, m.nv * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.xi, m.nv * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.h, m.nv * sizeof(float)));
	HandleError(hipMalloc((void**)&dm.d, m.nv * sizeof(float)));

	dm.ncolors = m.ncolors;
	dm.nbcolors = m.nbcolors;
	
	HandleError(hipMalloc((void**)&dm.ncolor_sides, m.ncolors  * sizeof(int)));
	HandleError(hipMalloc((void**)&dm.nbcolor_sides, m.nbcolors * sizeof(int)));

	HandleError(hipMalloc((void**)&dm.color_sides, m.ncolors  * sizeof(int*)));
	HandleError(hipMalloc((void**)&dm.bcolor_sides, m.nbcolors * sizeof(int*)));

	for (int i = 0; i < m.ncolors; ++i)
		HandleError(hipMalloc((void**)dm.color_sides + i, m.ncolor_sides[i] * sizeof(int)));

	for (int i = 0; i < m.nbcolors; ++i)
		HandleError(hipMalloc((void**)dm.bcolor_sides + i, m.nbcolor_sides[i] * sizeof(int)));
}

void free_device_mem(device_mesh dm)
{
	HandleError(hipFree(dm.a  ));
	HandleError(hipFree(dm.dl ));
	HandleError(hipFree(dm.dn ));
	HandleError(hipFree(dm.nx ));
	HandleError(hipFree(dm.ny ));
	HandleError(hipFree(dm.v1 ));
	HandleError(hipFree(dm.v2 ));
	HandleError(hipFree(dm.ibc));

	HandleError(hipFree(dm.px));
	HandleError(hipFree(dm.py));
	HandleError(hipFree(dm.ux));
	HandleError(hipFree(dm.uy));
	HandleError(hipFree(dm.xi));
	HandleError(hipFree(dm.h ));
	HandleError(hipFree(dm.d ));

	for (int i = 0; i < dm.ncolors; ++i)
		HandleError(hipFree(dm.color_sides  + i));
	for (int i = 0; i < dm.nbcolors; ++i)
		HandleError(hipFree(dm.bcolor_sides + i));

	HandleError(hipFree(dm.ncolor_sides ));
	HandleError(hipFree(dm.nbcolor_sides));

	HandleError(hipFree(dm.color_sides  ));
	HandleError(hipFree(dm.bcolor_sides ));
}

void copy_mesh_data(device_mesh& dm, mesh m)
{
	float *a   = new float[m.nv];
	float *dl  = new float[m.ns];
	float *dn  = new float[m.ns];
	float *nx  = new float[m.ns];
	float *ny  = new float[m.ns];
	float *nbx = new float[m.nbs];
	float *nby = new float[m.nbs];
	int *v1  = new int[m.ns];
	int *v2  = new int[m.ns];
	int *ibc = new int[m.ns];

	for (int i=0;i<m.nv;++i) a[i]=m.v[i].area;
	for (int i=0;i<m.ns;++i)
	{
		dl[i]  = m.s[i].dl;
		dn[i]  = m.s[i].dn;
		nx[i]  = m.s[i].n.x;
		ny[i]  = m.s[i].n.y;
		v1[i]  = m.s[i].v1->no;
		v2[i]  = m.s[i].v2->no;
		ibc[i] = m.s[i].ibc;
	}
	for (int i=0;i<m.nbs;++i)
	{
		nbx[i]=m.bsnormals[i].x;
		nby[i]=m.bsnormals[i].y;
	}

	HandleError(hipMemcpy(dm.a , a , m.nv * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.dl, dl, m.ns * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.dn, dn, m.ns * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.nx, nx, m.ns * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.ny, ny, m.ns * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.v1, v1, m.ns * sizeof(int  ), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.v2, v2, m.ns * sizeof(int  ), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.ibc, ibc, m.ns * sizeof(int  ), hipMemcpyHostToDevice));

	HandleError(hipMemcpy(dm.nbx, nbx, m.nbs * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.nby, nby, m.nbs * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.bsides  , m.bsides  , m.nbs * sizeof(int), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.bvertices, m.bvertices, m.nbv * sizeof(int), hipMemcpyHostToDevice));

	HandleError(hipMemcpy(dm.ncolor_sides, m.ncolor_sides, m.ncolors *sizeof(int), hipMemcpyHostToDevice));
	HandleError(hipMemcpy(dm.nbcolor_sides, m.nbcolor_sides, m.nbcolors*sizeof(int), hipMemcpyHostToDevice));

	for (int i = 0; i < dm.ncolors; ++i)
		HandleError(hipMemcpy(dm.color_sides[i], m.color_sides[i], m.ncolor_sides[i] * sizeof(int), hipMemcpyHostToDevice));
	for (int i = 0; i < dm.nbcolors; ++i)
		HandleError(hipMemcpy(dm.bcolor_sides[i], m.bcolor_sides[i], m.nbcolor_sides[i] * sizeof(int), hipMemcpyHostToDevice));

	delete[] a ;
	delete[] dl;
	delete[] dn;
	delete[] nx;
	delete[] ny;
	delete[] v1;
	delete[] v2;
	delete[] nbx;
	delete[] nby;
	delete[] ibc;
}

struct color_data
{
	int nsides;
	int* sides;
};

__global__ void _grad_sides(device_mesh m, color_data d, float *phi, float *gradx, float *grady)
{
	int icolor = blockDim.x * blockIdx.x + threadIdx.x;
	if (icolor >= d.nsides) return;
	int i = d.sides[icolor];

	int v1 = m.v1[i];
	int v2 = m.v2[i];

	float phif = ( phi[v1] + phi[v2] ) / 2;

	float gx= phif * m.nx[i] * m.dn[i];
	float gy= phif * m.ny[i] * m.dn[i];

	gradx[v1] += gx; grady[v1] += gy;
	gradx[v2] -= gx; grady[v2] -= gy;
}

__global__ void _grad_bsides(device_mesh m, color_data d, float *phi, float *gradx, float *grady)
{
	int icolor = blockDim.x * blockIdx.x + threadIdx.x;
	if (icolor >= d.nsides) return;
	int i = d.sides [icolor];
	int j = m.ibc[i];

	int v1 = m.v1[i];
	int v2 = m.v2[i];

	float phif1 = 0.75 * phi[v1] + 0.25 * phi[v2];
	float phif2 = 0.25 * phi[v1] + 0.75 * phi[v2];

	gradx[v1] += phif1 * m.nbx[j] * m.dl[i] /2; grady[v1] += phif1 * m.nbx[j] * m.dl[i] /2;
	gradx[v2] += phif2 * m.nbx[j] * m.dl[i] /2; grady[v2] += phif2 * m.nbx[j] * m.dl[i] /2;
}

__global__ void _devide_by_a(device_mesh m, float *phi)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= m.nv) return;

	phi[i] /= m.a[i];
}

__global__ void _multiply_by_a(device_mesh m, float *phi)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= m.nv) return;

	phi[i] *= m.a[i];
}

void get_grad(device_mesh m, float *phi, float*& gradx, float*& grady)
{
	HandleError( hipMalloc( (void**)&gradx, m.nv * sizeof(float)));
	HandleError( hipMalloc( (void**)&grady, m.nv * sizeof(float)));

	int nthreads = 512;
	color_data d;
	d.nsides=1;

	_grad_sides<<<m.ns/nthreads+1,nthreads>>>(m, d, phi, gradx, grady);
	_grad_bsides<<<m.nbs/nthreads+1,nthreads>>>(m, d, phi, gradx, grady);
	_devide_by_a<<<m.nv/nthreads+1,nthreads>>>(m, phi);
}

void copy_to_device(device_mesh dm, domain d)
{
	HandleError(hipMemcpy( dm.px, d.px, dm.nv * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy( dm.py, d.py, dm.nv * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy( dm.ux, d.ux, dm.nv * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy( dm.uy, d.uy, dm.nv * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy( dm.xi, d.xi, dm.nv * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy( dm.h , d.h , dm.nv * sizeof(float), hipMemcpyHostToDevice));
	HandleError(hipMemcpy( dm.d , d.d , dm.nv * sizeof(float), hipMemcpyHostToDevice));
}
void copy_to_host(device_mesh dm, domain d)
{
	HandleError(hipMemcpy( d.px, dm.px, dm.nv * sizeof(float), hipMemcpyDeviceToHost));
	HandleError(hipMemcpy( d.py, dm.py, dm.nv * sizeof(float), hipMemcpyDeviceToHost));
	HandleError(hipMemcpy( d.ux, dm.ux, dm.nv * sizeof(float), hipMemcpyDeviceToHost));
	HandleError(hipMemcpy( d.uy, dm.uy, dm.nv * sizeof(float), hipMemcpyDeviceToHost));
	HandleError(hipMemcpy( d.xi, dm.xi, dm.nv * sizeof(float), hipMemcpyDeviceToHost));
	HandleError(hipMemcpy( d.h , dm.h , dm.nv * sizeof(float), hipMemcpyDeviceToHost));
	// HandleError(hipMemcpy(d.m.d , dm.d , dm.nv * sizeof(float), hipMemcpyDeviceToHost));
}
